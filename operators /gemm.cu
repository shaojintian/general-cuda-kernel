#include <hip/hip_runtime.h>
#include <iostream>

#define TILE_SIZE 16

__global__ void gemm_kernel(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float value = 0.0f;
        for (int i = 0; i < K; i++) {
            value += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = value;
    }
}

extern "C" void gemm(float* A, float* B, float* C, int M, int N, int K) {
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);

    gemm_kernel<<<gridDim, blockDim>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}